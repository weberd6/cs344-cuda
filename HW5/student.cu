#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/

#include "utils.h"
#include <stdio.h>

__global__
void histo(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  extern __shared__ unsigned int s_histo[];
  int pos = blockIdx.x * blockDim.x + threadIdx.x;

  if (pos >= numVals)
      return;

  // Zero all shared
  s_histo[threadIdx.x] = 0;
  __syncthreads();

  atomicAdd(&s_histo[vals[pos]], 1);

  __syncthreads();

  atomicAdd(&histo[threadIdx.x], s_histo[threadIdx.x]);
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  
  const int threadsPerBlock = 1024;
  int numBlocks = ceil(((float)numElems)/threadsPerBlock);
  histo<<<numBlocks, threadsPerBlock, threadsPerBlock*sizeof(unsigned int)>>>
      (d_vals, d_histo, numElems);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

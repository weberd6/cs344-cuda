#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.


      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly -
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */

#include "utils.h"
#include <thrust/host_vector.h>
#include "reference_calc.cpp"

__global__
void computeMask(const uchar4* d_sourceImg,
                 unsigned char* d_mask,
                 const size_t numRowsSource,
                 const size_t numColsSource)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > numRowsSource*numColsSource)
        return;

    d_mask[i] = (d_sourceImg[i].x + d_sourceImg[i].y + d_sourceImg[i].z < 3 * 255) ? 1 : 0;
}

__global__
void interiorBorder(unsigned char* borderPixels,
                    unsigned char* strictInteriorPixels,
                    unsigned char* mask,
                    const size_t numRowsSource,
                    const size_t numColsSource)
{
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;

    if ((r == 0) || (r >= numRowsSource-1) || (c == 0) || (c >= numRowsSource-1))
        return;

    if (mask[r * numColsSource + c]) {
      if (mask[(r -1) * numColsSource + c] && mask[(r + 1) * numColsSource + c] &&
          mask[r * numColsSource + c - 1] && mask[r * numColsSource + c + 1]) {
        strictInteriorPixels[r * numColsSource + c] = 1;
        borderPixels[r * numColsSource + c] = 0;
        //interiorPixelList.push_back(make_uint2(r, c));
      }
      else {
        strictInteriorPixels[r * numColsSource + c] = 0;
        borderPixels[r * numColsSource + c] = 1;
      }
    }
    else {
        strictInteriorPixels[r * numColsSource + c] = 0;
        borderPixels[r * numColsSource + c] = 0;
    }
}

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{
    const uchar4* d_sourceImg;
    const uchar4* d_destImg;
    const uchar4* d_blendedImg;

    const int srcSize = numRowsSource*numColsSource;
    hipMalloc(&d_sourceImg, srcSize*sizeof(unsigned char));
    hipMalloc(&d_destImg, srcSize*sizeof(unsigned char));
    hipMalloc(&d_blendedImg, srcSize*sizeof(unsigned char));

  /* To Recap here are the steps you need to implement

     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied. */
    unsigned char* d_mask;
    hipMalloc(&d_mask, srcSize*sizeof(unsigned char));

    const int threadsPerBlock = 1024;
    int K = ceil(((float)srcSize)/threadsPerBlock);
    computeMask<<<K, threadsPerBlock>>>(d_sourceImg, d_mask, numRowsSource, numColsSource);

    /*
     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't. */
    unsigned char *d_borderPixels;
    unsigned char *d_strictInteriorPixels;
    hipMalloc(&d_borderPixels, srcSize*sizeof(unsigned char));
    hipMalloc(&d_strictInteriorPixels, srcSize*sizeof(unsigned char));

    const dim3 squareBlock(32, 32, 1);
    const dim3 squareGrid(ceil(((float)numColsSource)/32), ceil(((float)numRowsSource)/32), 1);
    interiorBorder<<<squareGrid, squareBlock>>>
        (d_borderPixels, d_strictInteriorPixels, d_mask, numRowsSource, numColsSource);

    /*
     3) Separate out the incoming image into three separate channels

     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.

     5) For each color channel perform the Jacobi iteration described
        above 800 times.

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.

      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */



  /* The reference calculation is provided below, feel free to use it
     for debugging purposes.
   */

  /*
    uchar4* h_reference = new uchar4[srcSize];
    reference_calc(h_sourceImg, numRowsSource, numColsSource,
                   h_destImg, h_reference);

    checkResultsEps((unsigned char *)h_reference, (unsigned char *)h_blendedImg, 4 * srcSize, 2, .01);
    delete[] h_reference; */
}

